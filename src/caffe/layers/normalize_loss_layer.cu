#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/normalize_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void NormalizeLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
      const Dtype* predict = bottom[0]->gpu_data();
      const Dtype* ground_truth = bottom[1]->gpu_data();
      const Dtype* visiable = bottom[2]->gpu_data();
      const Dtype* normalize_param = bottom[3]->gpu_data();
    
      int batch_size = bottom[0]->shape(0);
      int sample_size = bottom[0]->shape(1);
      int pts = sample_size / 2;
    
      Dtype total_error = 0.0;
      n_visiables_ = 0;
      for (int i = 0; i < batch_size; ++i) {
        for (int j = 0; j < pts; ++j) {
          Dtype p_x = predict[i * sample_size + j * 2];
          Dtype p_y = predict[i * sample_size + j * 2 + 1];
    
          Dtype g_x = ground_truth[i * sample_size + j * 2];
          Dtype g_y = ground_truth[i * sample_size + j * 2 + 1];
    
          Dtype v = visiable[i * pts + j];
          if (v <= 0.5) {
            // this keypoint is invisiable
            continue;
          }
          // calculate normalized distance
          Dtype dist = sqrtf((p_x - g_x) * (p_x - g_x) + (p_y - g_y) * (p_y - g_y));
          dist /= normalize_param[i];
    
          total_error += dist;
          n_visiables_++;
        }
      }
      top[0]->mutable_cpu_data()[0] = total_error / n_visiables_;
}

template <typename Dtype>
void NormalizeLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
      const Dtype* predict = bottom[0]->gpu_data();
      const Dtype* ground_truth = bottom[1]->gpu_data();
      const Dtype* visiable = bottom[2]->gpu_data();
      const Dtype* normalize_param = bottom[3]->gpu_data();
    
      int batch_size = bottom[0]->shape(0);
      int sample_size = bottom[0]->shape(1);
      int pts = sample_size / 2;
    
      // only calculate predict diff
      Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
      const Dtype top_diff = top[0]->gpu_diff()[0];
      for (int i = 0; i < batch_size; ++i) {
        Dtype K = this->n_visiables_ * normalize_param[i];
        for (int j = 0; j < pts; ++j) {
          Dtype p_x = predict[i * sample_size + j * 2];
          Dtype p_y = predict[i * sample_size + j * 2 + 1];
    
          Dtype g_x = ground_truth[i * sample_size + j * 2];
          Dtype g_y = ground_truth[i * sample_size + j * 2 + 1];
    
          Dtype v = visiable[i * pts + j];
          if (v <= 0.5) {
            // no diff
            bottom_diff[i * sample_size + j * 2] = 0;
            bottom_diff[i * sample_size + j * 2 + 1] = 0;
          } else {
            Dtype dist =
                sqrtf((p_x - g_x) * (p_x - g_x) + (p_y - g_y) * (p_y - g_y));
            bottom_diff[i * sample_size + j * 2] = p_x / dist / K * top_diff;
            bottom_diff[i * sample_size + j * 2 + 1] = p_y / dist / K * top_diff;
          }
        }
      }
}

INSTANTIATE_LAYER_GPU_FUNCS(NormalizeLossLayer);

}  // namespace caffe
